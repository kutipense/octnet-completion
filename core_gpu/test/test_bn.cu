#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <cstdlib>
#include <cstdio>
#include <cmath>

#include "octnet/core/core.h"
#include "octnet/gpu/gpu.h"
#include "octnet/test/objects.h"
#include "octnet/gpu/bn.h"

#define EPS 1e-4

inline void expect(bool should_be_true, const char* message) {
  if (!should_be_true) {
    printf("%s\n", message);
    exit(-1);
  }
}

void test_norm() {
  octree* grid_h = create_test_bn_octree_2x8x8x8x2_fixed();
  octree* grid_d = octree_new_gpu();
  octree_to_gpu(grid_h, grid_d);
  
  ot_data_t avg1 = 0;
  ot_data_t avg2 = 0;
  ot_data_t var1 = 0;
  ot_data_t var2 = 0;
  
  for (int i = 0; i < 60; i += 2) {
    ot_data_t factor = 1;
    if (i < 30) {
      factor = 4*4*4;
    }
    if (i >= 30 && i < 44) {
      factor = 2*2*2;
    }
    
    avg1 += factor*i;
    avg2 += factor*(i + 1);
    var1 += factor*i*i;
    var2 += factor*(i + 1)*(i + 1);
  }
  
  avg1 /= 2*8*8*8;
  avg2 /= 2*8*8*8;
  var1 = var1/(2*8*8*8) - avg1*avg1;
  var2 = var2/(2*8*8*8) - avg2*avg2;
  
  ot_data_t* avgs_h = new ot_data_t[2];
  ot_data_t* vars_h = new ot_data_t[2];
  avgs_h[0] = 0;
  avgs_h[1] = 0;
  vars_h[0] = 0;
  vars_h[1] = 0;
  
  ot_data_t* avgs_d; hipMalloc((ot_data_t**)&avgs_d, 2*sizeof(ot_data_t));
  ot_data_t* vars_d; hipMalloc((ot_data_t**)&vars_d, 2*sizeof(ot_data_t));
  hipMemcpy(avgs_d, avgs_h, 2*sizeof(ot_data_t), hipMemcpyHostToDevice);
  hipMemcpy(vars_d, vars_h, 2*sizeof(ot_data_t), hipMemcpyHostToDevice);
  
  octree* grid_norm_d = octree_new_gpu();
  octree_bn_norm_gpu(grid_d, avgs_d, vars_d, grid_norm_d);
  octree* grid_norm_h = octree_new_cpu();
  octree_to_cpu(grid_norm_d, grid_norm_h);
  
  hipMemcpy(avgs_h, avgs_d, 2*sizeof(ot_data_t), hipMemcpyDeviceToHost);
  hipMemcpy(vars_h, vars_d, 2*sizeof(ot_data_t), hipMemcpyDeviceToHost);
  
  char* buffer = new char[100];
  sprintf(buffer, "average 1 not right: %f != %f", avg1, avgs_h[0]);
  expect(fabs(avg1 - avgs_h[0]) < EPS, buffer);
  
  sprintf(buffer, "average 2 not right: %f != %f", avg2, avgs_h[1]);
  expect(fabs(avg2 - avgs_h[1]) < EPS, "average 2 not right");
  
  sprintf(buffer, "variance 1 not right: %f != %f", var1, vars_h[0]);
  expect(fabs(var1 - vars_h[0]) < EPS, "variance 1 not right");
  
  sprintf(buffer, "variance 2 not right: %f != %f", var2, vars_h[1]);
  expect(fabs(var2 - vars_h[1]) < EPS, "variance 2 not right");
//  printf("%f %f %f %f %f %f %f %f", avg1, avgs[0], avg2, avgs[1], var1, vars[0], var2, vars[1]);
  
  for(int idx = 0; idx < grid_h->n_leafs; ++idx) {
    for (int c = 0; c < 2; ++c) {
      ot_data_t val = (grid_h->data[idx*2 + c] - avgs_h[c])/sqrt(vars_h[c] + 1e-12);
      sprintf(buffer, "normalization not right (%d, %d): %f != %f", idx, c, grid_norm_h->data[idx*2 + c], val);
      expect(fabs(grid_norm_h->data[idx*2 + c] - val) < EPS, buffer);
    }
  }
  
  octree_free_cpu(grid_h);
  octree_free_cpu(grid_norm_h);
  octree_free_gpu(grid_d);
  octree_free_gpu(grid_norm_d);
  hipFree(avgs_d);
  hipFree(vars_d);
  delete[] avgs_h;
  delete[] vars_h;
  delete[] buffer;
}

// Not really good test as all the gradients should be zero.
void test_norm_bwd() {
  const ot_data_t one = 1.f;
  const ot_data_t two = 2.f;
  octree* grid_in_h = create_test_bn_octree_2x8x8x8x2_value(one);
  octree* grad_out_h = create_test_bn_octree_2x8x8x8x2_value(two);
  
  ot_data_t* avgs_h = new ot_data_t[2];
  ot_data_t* vars_h = new ot_data_t[2];
  avgs_h[0] = 5;
  avgs_h[1] = 0.5;
  vars_h[0] = 2.5;
  vars_h[1] = 0.5;
  
  ot_data_t* avgs_d; hipMalloc((ot_data_t**)&avgs_d, 2*sizeof(ot_data_t));
  ot_data_t* vars_d; hipMalloc((ot_data_t**)&vars_d, 2*sizeof(ot_data_t));
  hipMemcpy(avgs_d, avgs_h, 2*sizeof(ot_data_t), hipMemcpyHostToDevice);
  hipMemcpy(vars_d, vars_h, 2*sizeof(ot_data_t), hipMemcpyHostToDevice);
  
  octree* grid_in_d = octree_new_gpu();
  octree* grad_out_d = octree_new_gpu();
  octree* grad_in_d = octree_new_gpu();
  
  octree_to_gpu(grid_in_h, grid_in_d);
  octree_to_gpu(grad_out_h, grad_out_d);
  
  octree_bn_norm_bwd_gpu(grid_in_d, grad_out_d, avgs_d, vars_d, grad_in_d);
  octree* grad_in_h = octree_new_cpu();
  octree_to_cpu(grad_in_d, grad_in_h);

  hipMemcpy(avgs_h, avgs_d, 2*sizeof(ot_data_t), hipMemcpyDeviceToHost);
  hipMemcpy(vars_h, vars_d, 2*sizeof(ot_data_t), hipMemcpyDeviceToHost);
  
  ot_data_t* dl_davgs = new ot_data_t[2];
  ot_data_t* dl_dvars = new ot_data_t[2];
  ot_data_t* dl_dx = new ot_data_t[2];
  
  for (int c = 0; c < 2; ++c) {
    dl_dvars[c] = 2*8*8*8*two*(one - avgs_h[c])*-.5*pow(vars_h[c] + 1e-12, -1.5f);
    dl_davgs[c] = 2*8*8*8*two*(-1)/sqrt(vars_h[c] + 1e-12) + dl_dvars[c]*(-2)*(one - avgs_h[c]);
    dl_dx[c] = two/sqrt(vars_h[c] + 1e-12) + dl_dvars[c]*2*(one - avgs_h[c])/(2*8*8*8) + dl_davgs[c]/(2*8*8*8);
  }
  
  char* buffer = new char[100];
  for(int idx = 0; idx < grad_in_h->n_leafs; ++idx) {
    for (int c = 0; c < 2; ++c) {
      sprintf(buffer, "gradients not right (%d, %d): %f != %f", idx, c, grad_in_h->data[idx*2 + c], dl_dx[c]);
      expect(fabs(grad_in_h->data[idx*2 + c] - dl_dx[c]) < EPS, buffer);
    }
  }
  
  octree_free_cpu(grid_in_h);
  octree_free_cpu(grad_out_h);
  octree_free_cpu(grad_in_h);
  octree_free_gpu(grad_in_d);
  octree_free_gpu(grad_out_d);
  octree_free_gpu(grid_in_d);
  delete[] avgs_h;
  delete[] vars_h;
  delete[] dl_davgs;
  delete[] dl_dvars;
  delete[] dl_dx;
  hipFree(avgs_d);
  hipFree(vars_d);
}

int main(int argc, char** argv) {
  test_norm();
  test_norm_bwd();
  return 0;
}