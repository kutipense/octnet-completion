#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "octnet/gpu/conv.h"
#include "octnet/gpu/gpu.h"
#include "octnet/gpu/oc2col.h"
#include "octnet/gpu/col2oc.h"
#include "octnet/gpu/buffer.h"

#include <thrust/fill.h>
#include <thrust/execution_policy.h>


void print_matrix_gpu(const ot_data_t* data_d, int rows, int cols) {
  ot_data_t* data_h = device_to_host_malloc(data_d, rows*cols);
  int idx = 0;
  printf("[");
  for(int row = 0; row < rows; ++row) {
    if(row > 0) printf(" ");
    printf("[ ");
    for(int col = 0; col < cols; ++col) {
      printf("%f", data_h[idx]);
      idx++;
      if(col < cols-1) {
        printf(", ");
      }
    }
    if(row < rows - 1) {
      printf(" ], \n");
    }
    else {
      printf(" ]] \n");
    }
  }
  delete[] data_h;
}


__global__ void kernel_conv_mm_add_bias(ot_data_t* out, int n_leafs, int channels_out, const ot_data_t* bias) {
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    for(int f = 0; f < channels_out; ++f) {
      out[leaf_idx * channels_out + f] += bias[f];
    }
  }
}


void octree_conv_mm_gpu(hipblasHandle_t cublas_handle, const octree* in, const ot_data_t* weights, const ot_data_t* bias, int channels_out, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_gpu\n"); }

  octree_resize_gpu(in->n, in->grid_depth, in->grid_height, in->grid_width, channels_out, in->n_leafs, out);
  octree_cpy_scalars(in, out);
  out->feature_size = channels_out;
  octree_cpy_trees_cpu_gpu(in, out);
  octree_cpy_prefix_leafs_gpu_gpu(in, out);

  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(in->n_leafs * K333 * in->feature_size);

  oc2col_gpu(in, col_buffer.data(), col_buffer.capacity());

  float alpha = 1;
  float beta = 0;
  int m = out->feature_size;
  int n = in->n_leafs;
  int k = in->feature_size * K333;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_T, HIPBLAS_OP_N,
      m, n, k,
      &alpha,
      weights, k,
      col_buffer.data(), k,
      &beta,
      out->data, m
    )
  );

  // add bias
  kernel_conv_mm_add_bias<<<GET_BLOCKS(out->n_leafs), CUDA_NUM_THREADS>>>(
     out->data, out->n_leafs, out->feature_size, bias
  );
  CUDA_POST_KERNEL_CHECK;

  // printf("--------- col_buffer ----------\n");
  // print_matrix_gpu(col_buffer->data, in->n_leafs, K333*in->feature_size);
  // printf("--------- weights ----------\n");
  // print_matrix_gpu(weights, out->feature_size, K333*in->feature_size);
  // printf("--------- out ----------\n");
  // print_matrix_gpu(out->data, in->n_leafs, out->feature_size);
}


void octree_conv_mm_bwd_gpu(hipblasHandle_t cublas_handle, const octree* grad_out, const ot_data_t* weights, int channels_in, octree* grad_in) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_bwd_gpu\n"); }

  octree_resize_gpu(grad_out->n, grad_out->grid_depth, grad_out->grid_height, grad_out->grid_width, channels_in, grad_out->n_leafs, grad_in);
  octree_cpy_scalars(grad_out, grad_in);
  grad_in->feature_size = channels_in;
  octree_cpy_trees_cpu_gpu(grad_out, grad_in);
  octree_cpy_prefix_leafs_gpu_gpu(grad_out, grad_in);
  
  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(grad_in->n_leafs * K333 * channels_in);

  float alpha = 1;
  float beta = 0;
  int m = grad_in->feature_size * K333;
  int n = grad_in->n_leafs;
  int k = grad_out->feature_size;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      m, n, k,
      &alpha,
      weights, m,
      grad_out->data, k,
      &beta,
      col_buffer.data(), m
    )
  );

  col2oc_gpu(col_buffer.data(), grad_in);

  // printf("--------- weights ----------\n");
  // print_matrix_gpu(weights, grad_out->feature_size, K333*grad_in->feature_size);
  // printf("--------- grad_out ----------\n");
  // print_matrix_gpu(grad_out->data, grad_in->n_leafs, grad_out->feature_size);
  // printf("--------- col_buffer ----------\n");
  // print_matrix_gpu(col_buffer->data, grad_in->n_leafs, K333*grad_in->feature_size);
}

// __global__ void kernel_conv_mm_set(ot_data_t* out, int N, const ot_data_t val) {
//   CUDA_KERNEL_LOOP(idx, N) {
//     // if(idx >= N) printf("[ERROR] idx >= N in kernel_conv_mm_set\n");
//     out[idx] = val;
//   }
// }

void octree_conv_mm_wbwd_gpu(hipblasHandle_t cublas_handle, const octree* in, const octree* grad_out, const float scale, ot_data_t* grad_weights, ot_data_t* grad_bias) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_wbwd_gpu\n"); }

  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(in->n_leafs * K333 * in->feature_size);

  oc2col_gpu(in, col_buffer.data(), col_buffer.capacity());

  float alpha = scale;
  float beta = 1;
  int m = in->feature_size * K333;
  int n = grad_out->feature_size;
  int k = in->n_leafs;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      m, n, k,
      &alpha,
      col_buffer.data(), m,
      grad_out->data, n,
      &beta,
      grad_weights, m
    )
  );

  if(DEBUG > 1) { printf("[DEBUG] octree_conv_mm_wbwd_gpu kernel_conv_mm_set call col_buffer.data=%p, col_buffer.capacity=%d\n", col_buffer.data(), col_buffer.capacity()); }
  // kernel_conv_mm_set<<<GET_BLOCKS(in->n_leafs), CUDA_NUM_THREADS>>>(col_buffer->data, in->n_leafs, 1.f);
  // kernel_conv_mm_set<<<GET_BLOCKS_T(col_buffer->capacity, 512), 512>>>(col_buffer->data, col_buffer->capacity, 1.f);
  // CUDA_POST_KERNEL_CHECK;
  // thrust::fill(thrust::device, col_buffer->data, col_buffer->data + col_buffer->capacity, 1.f);
  thrust::fill(thrust::device, col_buffer.data(), col_buffer.data() + in->n_leafs, 1.f);

  alpha = scale;
  beta = 1;
  m = grad_out->feature_size;
  n = in->n_leafs;
  CUBLAS_CHECK(
    hipblasSgemv(
      cublas_handle,
      HIPBLAS_OP_N,
      m, n,
      &alpha,
      grad_out->data, m,
      col_buffer.data(), 1,
      &beta,
      grad_bias, 1
    )
  );

  // printf("--------- grad_out ----------\n");
  // print_matrix_gpu(grad_out->data, in->n_leafs, grad_out->feature_size);
  // printf("--------- col_buffer ----------\n");
  // print_matrix_gpu(col_buffer->data, in->n_leafs, 1);
  // printf("--------- grad_bias ----------\n");
  // print_matrix_gpu(grad_bias, grad_out->feature_size, 1);
}
