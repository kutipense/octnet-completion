#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "octnet/gpu/combine.h"
#include "octnet/gpu/gpu.h"

#include <cstdio>
#include <cstdlib>



__global__ void kernel_concat(ot_data_t* out, int n_leafs, const ot_data_t* in1, const ot_data_t* in2, const ot_size_t feature_size_in1, const ot_size_t feature_size_in2, const ot_size_t feature_size_out) {
  CUDA_KERNEL_LOOP(vx_idx, n_leafs) {
    octree_cpy_leaf(in1 + vx_idx * feature_size_in1, feature_size_in1, out + vx_idx * feature_size_out);
    octree_cpy_leaf(in2 + vx_idx * feature_size_in2, feature_size_in2, out + vx_idx * feature_size_out + feature_size_in1);
  }
}

void octree_concat_gpu(const octree* in1, const octree* in2, bool check, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_concat_gpu\n"); }

  if(check && (!octree_equal_trees_gpu(in1, in2))) {
    printf("ERROR: tree structure of inputs do not match\n");
    exit(-1);
  }

  ot_size_t feature_size_in1 = in1->feature_size;
  ot_size_t feature_size_in2 = in2->feature_size;
  ot_size_t feature_size_out = feature_size_in1 + feature_size_in2;

  octree_resize_gpu(in1->n, in1->grid_depth, in1->grid_height, in1->grid_width, feature_size_out, in1->n_leafs, out);
  octree_cpy_trees_gpu_gpu(in1, out);
  octree_cpy_prefix_leafs_gpu_gpu(in1, out);

  kernel_concat<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
      out->data, in1->n_leafs, in1->data, in2->data, feature_size_in1, feature_size_in2, feature_size_out
  );
  CUDA_POST_KERNEL_CHECK;
}


template <bool do_grad_in2>
__global__ void kernel_concat_bwd(ot_data_t* grad_in1, ot_data_t* grad_in2, int n_leafs, const ot_data_t* grad_out, const ot_size_t feature_size_in1, const ot_size_t feature_size_in2, const ot_size_t feature_size_out) {
  CUDA_KERNEL_LOOP(vx_idx, n_leafs) {
    octree_cpy_leaf(grad_out + vx_idx * feature_size_out, feature_size_in1, grad_in1 + vx_idx * feature_size_in1);
    if(do_grad_in2) {
      octree_cpy_leaf(grad_out + vx_idx * feature_size_out + feature_size_in1, feature_size_in2, grad_in2 + vx_idx * feature_size_in2);
    }
  }
}

void octree_concat_bwd_gpu(const octree* in1, const octree* in2, const octree* grad_out, bool do_grad_in2, octree* grad_in1, octree* grad_in2) {
  if(DEBUG) { printf("[DEBUG] octree_concat_bwd_gpu\n"); }

  octree_resize_as_gpu(in1, grad_in1);
  octree_cpy_trees_gpu_gpu(in1, grad_in1);
  octree_cpy_prefix_leafs_gpu_gpu(in1, grad_in1);
  
  octree_resize_as_gpu(in2, grad_in2);
  octree_cpy_trees_gpu_gpu(in2, grad_in2);
  octree_cpy_prefix_leafs_gpu_gpu(in2, grad_in2);

  ot_size_t feature_size_in1 = in1->feature_size;
  ot_size_t feature_size_in2 = in2->feature_size;
  ot_size_t feature_size_out = feature_size_in1 + feature_size_in2;

  if(do_grad_in2) {
    kernel_concat_bwd<true><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2->data, in1->n_leafs, grad_out->data, feature_size_in1, feature_size_in2, feature_size_out
    );
  }
  else {
    kernel_concat_bwd<false><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2->data, in1->n_leafs, grad_out->data, feature_size_in1, feature_size_in2, feature_size_out
    );

  }
  CUDA_POST_KERNEL_CHECK;
}







__global__ void kernel_concat_dense(ot_data_t* out, int n_leafs, const octree in1, const ot_data_t* in2, const ot_size_t feature_size1, const ot_size_t feature_size2, const ot_size_t feature_size_out) {
  const int dense_depth = 8 * in1.grid_depth;
  const int dense_height = 8 * in1.grid_height;
  const int dense_width = 8 * in1.grid_width;
  
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    octree_cpy_leaf(in1.data + leaf_idx * feature_size1, feature_size1, out + leaf_idx * feature_size_out);

    int grid_idx = leaf_idx_to_grid_idx(&in1, leaf_idx);
    const ot_tree_t* tree = octree_get_tree(&in1, grid_idx);

    int cum_n_leafs = in1.prefix_leafs[grid_idx];
    int data_idx = leaf_idx - cum_n_leafs;
    int bit_idx = data_idx_to_bit_idx(tree, data_idx);

    int n,ds,hs,ws;
    int depth = octree_ind_to_dense_ind(&in1, grid_idx, bit_idx, &n, &ds,&hs,&ws);
    int width = width_from_depth(depth);

    for(int f = 0; f < feature_size2; ++f) {
      ot_data_t val = 0;
      for(int d = ds; d < ds+width; ++d) {
      for(int h = hs; h < hs+width; ++h) {
      for(int w = ws; w < ws+width; ++w) {
        val += in2[(((n * feature_size2 + f) * dense_depth + d) * dense_height + h) * dense_width + w];
      }
      }
      }

      out[leaf_idx * feature_size_out + feature_size1 + f] = val / (width*width*width);
    }
  }
}

void octree_concat_dense_gpu(const octree* in1, const ot_data_t* in2, ot_size_t feature_size2, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_concat_dense_gpu\n"); }

  ot_size_t feature_size1 = in1->feature_size;
  ot_size_t feature_size_out = feature_size1 + feature_size2;

  octree_resize_gpu(in1->n, in1->grid_depth, in1->grid_height, in1->grid_width, feature_size_out, in1->n_leafs, out);
  octree_cpy_trees_gpu_gpu(in1, out);
  octree_cpy_prefix_leafs_gpu_gpu(in1, out);

  kernel_concat_dense<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
      out->data, in1->n_leafs, *in1, in2, feature_size1, feature_size2, feature_size_out
  );
  CUDA_POST_KERNEL_CHECK;
}


template <bool do_grad_in2>
__global__ void kernel_concat_dense_bwd(ot_data_t* grad_in1, ot_data_t* grad_in2, int n_leafs, const octree grad_out, const ot_size_t feature_size1, const ot_size_t feature_size2, const ot_size_t feature_size_out) {
  const int dense_depth = 8 * grad_out.grid_depth;
  const int dense_height = 8 * grad_out.grid_height;
  const int dense_width = 8 * grad_out.grid_width;

  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    octree_cpy_leaf(grad_out.data + leaf_idx * feature_size_out, feature_size1, grad_in1 + leaf_idx * feature_size1);

    if(do_grad_in2) {
      int grid_idx = leaf_idx_to_grid_idx(&grad_out, leaf_idx);
      const ot_tree_t* tree = octree_get_tree(&grad_out, grid_idx);

      int cum_n_leafs = grad_out.prefix_leafs[grid_idx];
      int data_idx = leaf_idx - cum_n_leafs;
      int bit_idx = data_idx_to_bit_idx(tree, data_idx);

      int n,ds,hs,ws;
      int depth = octree_ind_to_dense_ind(&grad_out, grid_idx, bit_idx, &n, &ds,&hs,&ws);
      int width = width_from_depth(depth);

      for(int f = 0; f < feature_size2; ++f) {
        ot_data_t val = grad_out.data[leaf_idx * grad_out.feature_size + feature_size1 + f];
        for(int d = ds; d < ds+width; ++d) {
        for(int h = hs; h < hs+width; ++h) {
        for(int w = ws; w < ws+width; ++w) {
          grad_in2[(((n * feature_size2 + f) * dense_depth + d) * dense_height + h) * dense_width + w] = val;
        }
        }
        }
      }
    }
  }
}

void octree_concat_dense_bwd_gpu(const octree* in1, const ot_data_t* in2, ot_size_t feature_size2, const octree* grad_out, bool do_grad_in2, octree* grad_in1, ot_data_t* grad_in2) {
  if(DEBUG) { printf("[DEBUG] octree_concat_dense_bwd_gpu\n"); }

  octree_resize_as_gpu(in1, grad_in1);
  octree_cpy_trees_gpu_gpu(in1, grad_in1);
  octree_cpy_prefix_leafs_gpu_gpu(in1, grad_in1);
  
  ot_size_t feature_size1 = in1->feature_size;
  ot_size_t feature_size_out = feature_size1 + feature_size2;

  if(do_grad_in2) {
    kernel_concat_dense_bwd<true><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2, in1->n_leafs, *grad_out, feature_size1, feature_size2, feature_size_out
    );
  }
  else {
    kernel_concat_dense_bwd<false><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2, in1->n_leafs, *grad_out, feature_size1, feature_size2, feature_size_out
    );

  }
  CUDA_POST_KERNEL_CHECK;
}
