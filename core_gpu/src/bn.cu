#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "octnet/gpu/bn.h"
#include "octnet/gpu/gpu.h"

#define FAST_POW(x, y) pow(x, y)
#define FAST_SQRT(x) sqrt(x)
#define EPS 1e-12

// TODO in principle parallelization could also include the channel!
__global__ void kernel_bn_stat(const octree grid, const ot_size_t n_blocks, ot_data_t* avgs, ot_data_t* vars) {
  const ot_size_t channels = grid.feature_size;
  
  CUDA_KERNEL_LOOP(grid_idx, n_blocks) {
    ot_tree_t* tree = octree_get_tree(&grid, grid_idx);
    ot_data_t* in_data = octree_get_data(&grid, grid_idx);
    
    // check L0 split:
    if(!tree_isset_bit(tree, 0)) {
      ot_data_t factor = 8*8*8;
      for (int c = 0; c < channels; ++c) {
        ot_data_t val = in_data[c];
        ot_data_t fval = factor*val;
        atomicAdd(&avgs[c], fval);
        atomicAdd(&vars[c], fval*val);
      }
    }
    else {

      int bit_idx_l1 = 1;
      for(int bdl1 = 0; bdl1 < 2; ++bdl1) {
        for(int bhl1 = 0; bhl1 < 2; ++bhl1) {
          for(int bwl1 = 0; bwl1 < 2; ++bwl1) {
            
            // check L1 split:
            if(!tree_isset_bit(tree, bit_idx_l1)) {
              int data_idx = tree_data_idx(tree, bit_idx_l1, channels);
              ot_data_t factor = 4*4*4;
              for (int c = 0; c < channels; ++c) {
                ot_data_t val = (in_data + data_idx)[c];
                ot_data_t fval = factor*val;
                atomicAdd(&avgs[c], fval);
                atomicAdd(&vars[c], fval*val);
              }
            }
            else {

              int bit_idx_l2 = tree_child_bit_idx(bit_idx_l1);
              for(int bdl2 = 0; bdl2 < 2; ++bdl2) {
                for(int bhl2 = 0; bhl2 < 2; ++bhl2) {
                  for(int bwl2 = 0; bwl2 < 2; ++bwl2) {
                    
                    // check L2 split:
                    if(!tree_isset_bit(tree, bit_idx_l2)) {
                      int data_idx = tree_data_idx(tree, bit_idx_l2, channels);
                      ot_data_t factor = 2*2*2;
                      for (int c = 0; c < channels; ++c) {
                        ot_data_t val = (in_data + data_idx)[c];
                        ot_data_t fval = factor*val;
                        atomicAdd(&avgs[c], fval);
                        atomicAdd(&vars[c], fval*val);
                      }
                    }
                    else {

                      int bit_idx_l3 = tree_child_bit_idx(bit_idx_l2);
                      for(int bdl3 = 0; bdl3 < 2; ++bdl3) {
                        for(int bhl3 = 0; bhl3 < 2; ++bhl3) {
                          for(int bwl3 = 0; bwl3 < 2; ++bwl3) {
                            int data_idx = tree_data_idx(tree, bit_idx_l3, channels);
                            for (int c = 0; c < channels; ++c) {
                              ot_data_t val = (in_data + data_idx)[c];
                              atomicAdd(&avgs[c], val);
                              atomicAdd(&vars[c], val*val);
                            }
                            
                            bit_idx_l3++;
                          }
                        }
                      }

                    }
                    
                    bit_idx_l2++;
                  }
                }
              } 

            } // else L1
            
            bit_idx_l1++;
          } // for bwl1
        } // for bhl1
      } // for bdl1
    } // else L0
  }
}

__global__ void kernel_bn_stat_norm(const ot_size_t M, ot_data_t* avgs, ot_data_t* vars) {
  const int c = threadIdx.x;
  avgs[c] /= M;
  vars[c] /= M;
  vars[c] -= avgs[c]*avgs[c];
}

__global__ void kernel_bn_norm(const octree grid_in, const ot_size_t n_blocks, ot_data_t* avgs, ot_data_t* vars, octree grid) {
  const ot_size_t channels = grid.feature_size;
  
  CUDA_KERNEL_LOOP(grid_idx, n_blocks) {
    ot_tree_t* tree = octree_get_tree(&grid_in, grid_idx);
    ot_data_t* in_data = octree_get_data(&grid_in, grid_idx);
    ot_data_t* out_data = octree_get_data(&grid, grid_idx);
    
    // check L0 split:
    if(!tree_isset_bit(tree, 0)) {
      for (int c = 0; c < channels; ++c) {
        out_data[c] = (in_data[c] - avgs[c])/FAST_SQRT(vars[c] + EPS);
      }
    }
    else {

      int bit_idx_l1 = 1;
      for(int bdl1 = 0; bdl1 < 2; ++bdl1) {
        for(int bhl1 = 0; bhl1 < 2; ++bhl1) {
          for(int bwl1 = 0; bwl1 < 2; ++bwl1) {
            
            // check L1 split:
            if(!tree_isset_bit(tree, bit_idx_l1)) {
              int data_idx = tree_data_idx(tree, bit_idx_l1, channels);
              for (int c = 0; c < channels; ++c) {
                out_data[data_idx + c] = (in_data[data_idx + c] - avgs[c])/FAST_SQRT(vars[c] + EPS);
              }
            }
            else {

              int bit_idx_l2 = tree_child_bit_idx(bit_idx_l1);
              for(int bdl2 = 0; bdl2 < 2; ++bdl2) {
                for(int bhl2 = 0; bhl2 < 2; ++bhl2) {
                  for(int bwl2 = 0; bwl2 < 2; ++bwl2) {
                    
                    // check L2 split:
                    if(!tree_isset_bit(tree, bit_idx_l2)) {
                      int data_idx = tree_data_idx(tree, bit_idx_l2, channels);
                      for (int c = 0; c < channels; ++c) {
                        out_data[data_idx + c] = (in_data[data_idx + c] - avgs[c])/FAST_SQRT(vars[c] + EPS);
                      }
                    }
                    else {

                      int bit_idx_l3 = tree_child_bit_idx(bit_idx_l2);
                      for(int bdl3 = 0; bdl3 < 2; ++bdl3) {
                        for(int bhl3 = 0; bhl3 < 2; ++bhl3) {
                          for(int bwl3 = 0; bwl3 < 2; ++bwl3) {
                            int data_idx = tree_data_idx(tree, bit_idx_l3, channels);
                            for (int c = 0; c < channels; ++c) {
                              out_data[data_idx + c] = (in_data[data_idx + c] - avgs[c])/FAST_SQRT(vars[c] + EPS);
                            }
                            
                            bit_idx_l3++;
                          }
                        }
                      }

                    }
                    
                    bit_idx_l2++;
                  }
                }
              } 

            } // else L1
            
            bit_idx_l1++;
          } // for bwl1
        } // for bhl1
      } // for bdl1
    } // else L0
  }
}

void octree_bn_norm_gpu(const octree* grid_in, ot_data_t* avgs, ot_data_t* vars, octree* grid) {
  octree_resize_gpu(grid_in->n, grid_in->grid_depth, grid_in->grid_height, grid_in->grid_width, grid_in->feature_size, grid_in->n_leafs, grid);
  octree_cpy_scalars(grid_in, grid);
  octree_cpy_trees_gpu_gpu(grid_in, grid);
  octree_cpy_prefix_leafs_gpu_gpu(grid_in, grid);
  octree_fill_data_gpu(grid, 0);
  
  const ot_size_t n_blocks = octree_num_blocks(grid_in);
  const ot_size_t channels = grid_in->feature_size;
  const ot_size_t M = 8*grid->grid_depth*8*grid->grid_height*8*grid->grid_width*grid->n;

  kernel_bn_stat<<<GET_BLOCKS_T(n_blocks, 512), 512>>>(*grid_in, n_blocks, avgs, vars);
  CUDA_POST_KERNEL_CHECK;
  
  kernel_bn_stat_norm<<<1, channels>>>(M, avgs, vars);
  CUDA_POST_KERNEL_CHECK;
  
  kernel_bn_norm<<<GET_BLOCKS_T(n_blocks, 512), 512>>>(*grid_in, n_blocks, avgs, vars, *grid);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void kernel_bn_ss(const octree grid_in, ot_size_t n_data, ot_data_t* gamma, ot_data_t *beta, octree grid_out) {
  CUDA_KERNEL_LOOP(data_idx, n_data) {
    ot_data_t val = grid_in.data[data_idx];
    ot_size_t c = data_idx%grid_in.feature_size;
    grid_out.data[data_idx] = gamma[c]*val + beta[c];
  }
}

void octree_bn_ss_gpu(const octree* grid_in, ot_data_t *gamma, ot_data_t *beta, bool inplace, octree* grid_out) {
  if (!inplace) {
    octree_resize_as_gpu(grid_in, grid_out);
    octree_cpy_scalars(grid_in, grid_out);
    octree_cpy_trees_gpu_gpu(grid_in, grid_out);
    octree_cpy_prefix_leafs_gpu_gpu(grid_in, grid_out);
  }
  
  const ot_size_t n_data = grid_in->n_leafs*grid_in->feature_size;
  kernel_bn_ss<<<GET_BLOCKS(n_data), CUDA_NUM_THREADS>>>(*grid_in, n_data, gamma, beta, *grid_out);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void kernel_bn_stat_bwd(const octree grid_in, const octree grad_out, const ot_size_t n_blocks, ot_data_t* avgs, ot_data_t* vars, ot_data_t* grad_avgs, ot_data_t* grad_avgs_part, ot_data_t* grad_vars) {
  const ot_size_t channels = grid_in.feature_size;
  
  CUDA_KERNEL_LOOP(grid_idx, n_blocks) {
    ot_tree_t* tree = octree_get_tree(&grid_in, grid_idx);
    ot_data_t* in_data = octree_get_data(&grid_in, grid_idx);
    ot_data_t* grad_out_data = octree_get_data(&grad_out, grid_idx);
    
    // check L0 split:
    if(!tree_isset_bit(tree, 0)) {
      ot_data_t factor = 8*8*8;
      for (int c = 0; c < channels; ++c) {
        ot_data_t grad = grad_out_data[c];
        ot_data_t val = in_data[c];
        ot_data_t centered = factor*(val - avgs[c]);
        atomicAdd(&grad_avgs[c], factor*grad);
        atomicAdd(&grad_avgs_part[c], centered);
        atomicAdd(&grad_vars[c], grad*centered);
      }
    }
    else {

      int bit_idx_l1 = 1;
      for(int bdl1 = 0; bdl1 < 2; ++bdl1) {
        for(int bhl1 = 0; bhl1 < 2; ++bhl1) {
          for(int bwl1 = 0; bwl1 < 2; ++bwl1) {
            
            // check L1 split:
            if(!tree_isset_bit(tree, bit_idx_l1)) {
              int data_idx = tree_data_idx(tree, bit_idx_l1, channels);
              ot_data_t factor = 4*4*4;
              for (int c = 0; c < channels; ++c) {
                ot_data_t grad = grad_out_data[data_idx + c];
                ot_data_t val = in_data[data_idx + c];
                ot_data_t centered = factor*(val - avgs[c]);
                atomicAdd(&grad_avgs[c], factor*grad);
                atomicAdd(&grad_avgs_part[c], centered);
                atomicAdd(&grad_vars[c], grad*centered);
              }
            }
            else {

              int bit_idx_l2 = tree_child_bit_idx(bit_idx_l1);
              for(int bdl2 = 0; bdl2 < 2; ++bdl2) {
                for(int bhl2 = 0; bhl2 < 2; ++bhl2) {
                  for(int bwl2 = 0; bwl2 < 2; ++bwl2) {
                    
                    // check L2 split:
                    if(!tree_isset_bit(tree, bit_idx_l2)) {
                      int data_idx = tree_data_idx(tree, bit_idx_l2, channels);
                      ot_data_t factor = 2*2*2;
                      for (int c = 0; c < channels; ++c) {
                        ot_data_t grad = grad_out_data[data_idx + c];
                        ot_data_t val = in_data[data_idx + c];
                        ot_data_t centered = factor*(val - avgs[c]);
                        atomicAdd(&grad_avgs[c], factor*grad);
                        atomicAdd(&grad_avgs_part[c], centered);
                        atomicAdd(&grad_vars[c], grad*centered);
                      }
                    }
                    else {

                      int bit_idx_l3 = tree_child_bit_idx(bit_idx_l2);
                      for(int bdl3 = 0; bdl3 < 2; ++bdl3) {
                        for(int bhl3 = 0; bhl3 < 2; ++bhl3) {
                          for(int bwl3 = 0; bwl3 < 2; ++bwl3) {
                            int data_idx = tree_data_idx(tree, bit_idx_l3, channels);
                            for (int c = 0; c < channels; ++c) {
                              ot_data_t grad = grad_out_data[data_idx + c];
                              ot_data_t val = in_data[data_idx + c];
                              ot_data_t centered = (val - avgs[c]);
                              atomicAdd(&grad_avgs[c], grad);
                              atomicAdd(&grad_avgs_part[c], centered);
                              atomicAdd(&grad_vars[c], grad*centered);
                            }
                            
                            bit_idx_l3++;
                          }
                        }
                      }

                    }
                    
                    bit_idx_l2++;
                  }
                }
              } 

            } // else L1
            
            bit_idx_l1++;
          } // for bwl1
        } // for bhl1
      } // for bdl1
    } // else L0
  } // for grid_idx
}

__global__ void kernel_bn_stat_norm_bwd(const ot_size_t M, ot_data_t* vars, ot_data_t* grad_avgs, ot_data_t* grad_avgs_part, ot_data_t* grad_vars) {
  const ot_size_t c = threadIdx.x;
  grad_vars[c] *= -0.5f*FAST_POW(vars[c] + EPS, -1.5f);
  grad_avgs[c] *= -1.f/FAST_SQRT(vars[c] + EPS);
  grad_avgs[c] += grad_vars[c]/M*(-2.f)*grad_avgs_part[c];
}

__global__ void kernel_bn_norm_init_bwd(const ot_size_t M, ot_data_t* vars, ot_data_t* grad_avgs, ot_data_t* grad_vars, ot_data_t* over_vars_eps, ot_data_t* grad_vars_over_M, ot_data_t* grad_avgs_over_M) {
  const ot_size_t c = threadIdx.x;
  over_vars_eps[c] = 1.f/FAST_SQRT(vars[c] + EPS);
  grad_vars_over_M[c] = grad_vars[c]*2.f/M;
  grad_avgs_over_M[c] = grad_avgs[c]/M;
}

__global__ void kernel_bn_norm_bwd(const octree grid_in, const octree grad_out, const ot_size_t n_blocks, ot_data_t* avgs, ot_data_t* grad_avgs_over_M, ot_data_t* grad_vars_over_M, ot_data_t* over_vars_eps, octree grad_in) {
  const ot_size_t channels = grid_in.feature_size;
  
  CUDA_KERNEL_LOOP(grid_idx, n_blocks) {
    ot_tree_t* tree = octree_get_tree(&grid_in, grid_idx);
    ot_data_t* in_data = octree_get_data(&grid_in, grid_idx);
    ot_data_t* grad_out_data = octree_get_data(&grad_out, grid_idx);
    ot_data_t* grad_in_data = octree_get_data(&grad_in, grid_idx);
    
    // check L0 split:
    if(!tree_isset_bit(tree, 0)) {
      for (int c = 0; c < channels; ++c) {
        grad_in_data[c] = grad_out_data[c]*over_vars_eps[c] 
            + grad_vars_over_M[c]*(in_data[c] - avgs[c]) + grad_avgs_over_M[c];
      }
    }
    else {

      int bit_idx_l1 = 1;
      for(int bdl1 = 0; bdl1 < 2; ++bdl1) {
        for(int bhl1 = 0; bhl1 < 2; ++bhl1) {
          for(int bwl1 = 0; bwl1 < 2; ++bwl1) {
            
            // check L1 split:
            if(!tree_isset_bit(tree, bit_idx_l1)) {
              int data_idx = tree_data_idx(tree, bit_idx_l1, channels);
              for (int c = 0; c < channels; ++c) {
                grad_in_data[data_idx + c] = grad_out_data[data_idx + c]*over_vars_eps[c] 
                    + grad_vars_over_M[c]*(in_data[data_idx + c] - avgs[c]) + grad_avgs_over_M[c];
              }
            }
            else {

              int bit_idx_l2 = tree_child_bit_idx(bit_idx_l1);
              for(int bdl2 = 0; bdl2 < 2; ++bdl2) {
                for(int bhl2 = 0; bhl2 < 2; ++bhl2) {
                  for(int bwl2 = 0; bwl2 < 2; ++bwl2) {
                    
                    // check L2 split:
                    if(!tree_isset_bit(tree, bit_idx_l2)) {
                      int data_idx = tree_data_idx(tree, bit_idx_l2, channels);
                      for (int c = 0; c < channels; ++c) {
                        grad_in_data[data_idx + c] = grad_out_data[data_idx + c]*over_vars_eps[c] 
                            + grad_vars_over_M[c]*(in_data[data_idx + c] - avgs[c]) + grad_avgs_over_M[c];
                      }
                    }
                    else {

                      int bit_idx_l3 = tree_child_bit_idx(bit_idx_l2);
                      for(int bdl3 = 0; bdl3 < 2; ++bdl3) {
                        for(int bhl3 = 0; bhl3 < 2; ++bhl3) {
                          for(int bwl3 = 0; bwl3 < 2; ++bwl3) {
                            int data_idx = tree_data_idx(tree, bit_idx_l3, channels);
                            for (int c = 0; c < channels; ++c) {
                              grad_in_data[data_idx + c] = grad_out_data[data_idx + c]*over_vars_eps[c] 
                                  + grad_vars_over_M[c]*(in_data[data_idx + c] - avgs[c]) + grad_avgs_over_M[c];
                            }
                            
                            bit_idx_l3++;
                          }
                        }
                      }

                    }
                    
                    bit_idx_l2++;
                  }
                }
              } 

            } // else L1
            
            bit_idx_l1++;
          } // for bwl1
        } // for bhl1
      } // for bdl1
    } // else L0
  }
}

void octree_bn_norm_bwd_gpu(const octree* grid_in, const octree* grad_out, ot_data_t* avgs, ot_data_t* vars, octree* grad_in) {
  octree_resize_gpu(grad_out->n, grad_out->grid_depth, grad_out->grid_height, grad_out->grid_width, grad_out->feature_size, grad_out->n_leafs, grad_in);
  octree_cpy_scalars(grad_out, grad_in);
  octree_cpy_trees_gpu_gpu(grad_out, grad_in);
  octree_cpy_prefix_leafs_gpu_gpu(grad_out, grad_in);
  octree_fill_data_gpu(grad_in, 0);
  
  const ot_size_t n_blocks = octree_num_blocks(grid_in);
  const ot_size_t channels = grid_in->feature_size;
  
  // Alloc arrays for gradients wrt average/variance directly on GPU:
  ot_data_t* grad_avgs; DEVICE_MALLOC(grad_avgs, channels); 
  ot_data_t* grad_avgs_part; DEVICE_MALLOC(grad_avgs_part, channels); 
  ot_data_t* grad_vars; DEVICE_MALLOC(grad_vars, channels); 
  
  DEVICE_MEMSET(grad_avgs, 0, channels);
  DEVICE_MEMSET(grad_avgs_part, 0, channels);
  DEVICE_MEMSET(grad_vars, 0, channels);
  
  kernel_bn_stat_bwd<<<GET_BLOCKS_T(n_blocks, 512), 512>>>(*grid_in, *grad_out, n_blocks, avgs, vars, grad_avgs, grad_avgs_part, grad_vars);
  CUDA_POST_KERNEL_CHECK;
//  printf("octree_bn_norm_bwd_gpu kernel_bn_stat_bwd finished\n");
  
  const ot_size_t M = 8*grid_in->grid_depth*8*grid_in->grid_height*8*grid_in->grid_width*grid_in->n;
  kernel_bn_stat_norm_bwd<<<1, channels>>>(M, vars, grad_avgs, grad_avgs_part, grad_vars);
  CUDA_POST_KERNEL_CHECK;
//  printf("octree_bn_norm_bwd_gpu kernel_bn_stat_norm_bwd finished\n");
   
  // Alloc arrays for helper variables to reduce computation directly on GPU:
  ot_data_t* over_vars_eps; DEVICE_MALLOC(over_vars_eps, channels);
  ot_data_t* grad_avgs_over_M; DEVICE_MALLOC(grad_avgs_over_M, channels);
  ot_data_t* grad_vars_over_M; DEVICE_MALLOC(grad_vars_over_M, channels);
  
//  DEVICE_MEMSET(over_vars_eps, 0, channels);
//  DEVICE_MEMSET(grad_avgs_over_M, 0, channels);
//  DEVICE_MEMSET(grad_vars_over_M, 0, channels);
  
  kernel_bn_norm_init_bwd<<<1, channels>>>(M, vars, grad_avgs, grad_vars, over_vars_eps, grad_vars_over_M, grad_avgs_over_M);
  CUDA_POST_KERNEL_CHECK;
//  printf("octree_bn_norm_bwd_gpu kernel_bn_norm_init_bwd finished\n");
  
  kernel_bn_norm_bwd<<<GET_BLOCKS_T(n_blocks, 512), 512>>>(*grid_in, *grad_out, n_blocks, avgs, grad_avgs_over_M, grad_vars_over_M, over_vars_eps, *grad_in);
  CUDA_POST_KERNEL_CHECK;
//  printf("octree_bn_norm_bwd_gpu kernel_bn_norm_bwd finished\n");
  
  DEVICE_FREE(grad_avgs);
  DEVICE_FREE(grad_avgs_part);
  DEVICE_FREE(grad_vars);
  DEVICE_FREE(over_vars_eps);
  DEVICE_FREE(grad_avgs_over_M);
  DEVICE_FREE(grad_vars_over_M);
}
__global__ void kernel_bn_ss_bwd(const octree grad_out, ot_size_t n_data, ot_data_t* gamma, octree grad_in) {
  CUDA_KERNEL_LOOP(data_idx, n_data) {
    ot_data_t val = grad_out.data[data_idx];
    ot_size_t c = data_idx%grad_out.feature_size;
    grad_in.data[data_idx] = gamma[c]*val;
  }
}

void octree_bn_ss_bwd_gpu(const octree* grad_out, ot_data_t* gamma, bool inplace, octree* grad_in) {
  if (!inplace) {
    octree_resize_as_gpu(grad_out, grad_in);
    octree_cpy_scalars(grad_out, grad_in);
    octree_cpy_trees_gpu_gpu(grad_out, grad_in);
    octree_cpy_prefix_leafs_gpu_gpu(grad_out, grad_in);
  }

  const ot_size_t n_data = grad_out->n_leafs*grad_out->feature_size;
  kernel_bn_ss_bwd<<<GET_BLOCKS(n_data), CUDA_NUM_THREADS>>>(*grad_out, n_data, gamma, *grad_in);
}

__global__ void kernel_bn_ss_wbwd(const octree grid_in, const octree grad_out, ot_size_t n_data, ot_data_t* grad_gamma, ot_data_t* grad_beta) {
  CUDA_KERNEL_LOOP(data_idx, n_data) {
    ot_data_t grad = grad_out.data[data_idx];
    ot_data_t val = grid_in.data[data_idx];
    ot_size_t c = data_idx%grid_in.feature_size;
    atomicAdd(&grad_gamma[c], grad*val);
    atomicAdd(&grad_beta[c], grad);
  }
}

void octree_bn_ss_wbwd_gpu(const octree* grid_in, const octree* grad_out, ot_data_t* grad_gamma, ot_data_t* grad_beta) {
  const ot_size_t n_data = grid_in->n_leafs*grid_in->feature_size;  
  
  kernel_bn_ss_wbwd<<<GET_BLOCKS(n_data), CUDA_NUM_THREADS>>>(*grid_in, *grad_out, n_data, grad_gamma, grad_beta);
  CUDA_POST_KERNEL_CHECK;
}